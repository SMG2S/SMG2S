#include "d_spgemm_d.h"

// error check macros
#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

int csr2d_csr(csrMatrix<double,int> A, d_csrMatrix<double,int> *B){

	B->d_row = A.row;
	B->d_column = A.column;
	B->d_nnz = A.nnz;

	hipMalloc(&B->d_csrRowPtr, (B->d_row+1)*sizeof(int));
        hipMalloc(&B->d_csrColInd, (B->d_nnz)*sizeof(int));
        hipMalloc(&B->d_csrVal, (B->d_nnz)*sizeof(double));

	hipMemcpy(B->d_csrRowPtr, A.csrRowPtr, (B->d_row+1)*sizeof(int), hipMemcpyHostToDevice);
  	hipMemcpy(B->d_csrColInd, A.csrColInd, B->d_nnz*sizeof(int), hipMemcpyHostToDevice);
  	hipMemcpy(B->d_csrVal, A.csrVal, B->d_nnz*sizeof(double), hipMemcpyHostToDevice);

	return 0;
}

int d_csr2csr(d_csrMatrix<double,int> B, csrMatrix<double,int> *A){

	A->row = B.d_row;
	A->column = B.d_column;
	A->nnz = B.d_nnz;

	A->csrRowPtr = (int *)malloc((B.d_row+1)*sizeof(int));
	A->csrColInd = (int *)malloc(B.d_nnz *sizeof(int));
	A->csrVal  = (double *)malloc(B.d_nnz *sizeof(double));

	hipMemcpy(A->csrRowPtr, B.d_csrRowPtr, (B.d_row+1)*sizeof(int), hipMemcpyDeviceToHost);
  	hipMemcpy(A->csrColInd, B.d_csrColInd,  B.d_nnz*sizeof(int), hipMemcpyDeviceToHost);
  	hipMemcpy(A->csrVal, B.d_csrVal, B.d_nnz*sizeof(double), hipMemcpyDeviceToHost);
  	cudaCheckErrors("hipMemcpy fail");

	return 0;
}

int d_spgemm_d(d_csrMatrix<double,int> A, d_csrMatrix<double,int> B, d_csrMatrix<double,int> *C, hipsparseHandle_t hndl)
{
	int baseC;
  	int *nnzTotalDevHostPtr = &C->d_nnz;
  	hipsparseMatDescr_t descrA, descrB, descrC;
  	hipsparseStatus_t stat;
  	CUSPARSE_CHECK(hipsparseCreate(&hndl));
  	stat = hipsparseCreateMatDescr(&descrA);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseCreateMatDescr(&descrB);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseCreateMatDescr(&descrC);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
  	CUSPARSE_CHECK(stat);
  	stat = hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
  	CUSPARSE_CHECK(stat);

  	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  	hipsparseOperation_t transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  	stat = hipsparseSetPointerMode(hndl, HIPSPARSE_POINTER_MODE_HOST);

  	CUSPARSE_CHECK(stat);
  	hipMalloc((void**)&C->d_csrRowPtr, sizeof(int)*(A.d_row+1));

  	cudaCheckErrors("hipMalloc fail");

  	C->d_row = A.d_row;
  	C->d_column = B.d_column;
  	stat = hipsparseXcsrgemmNnz(hndl, transA, transB, A.d_row, B.d_column, A.d_column,
        	descrA, A.d_nnz, A.d_csrRowPtr, A.d_csrColInd,
        	descrB, B.d_nnz, B.d_csrRowPtr, B.d_csrColInd,
        	descrC, C->d_csrRowPtr, nnzTotalDevHostPtr );

  	CUSPARSE_CHECK(stat);
	if (NULL != nnzTotalDevHostPtr){
    	C->d_nnz = *nnzTotalDevHostPtr;}
  	else{
    		hipMemcpy(&C->d_nnz, C->d_csrRowPtr+A.d_row, sizeof(int), hipMemcpyDeviceToHost);
    		hipMemcpy(&baseC, C->d_csrRowPtr, sizeof(int), hipMemcpyDeviceToHost);
    		cudaCheckErrors("hipMemcpy fail");
    		C->d_nnz -= baseC;
	}

	//  cudaCheckErrors("hipMalloc fail");
  	hipMalloc((void**)&C->d_csrColInd, sizeof(int)*C->d_nnz);
  	cudaCheckErrors("hipMalloc fail");
  	hipMalloc((void**)&C->d_csrVal, sizeof(double)*C->d_nnz);
  	cudaCheckErrors("hipMalloc fail");
	// perform multiplication C = A*B
  	stat = hipsparseDcsrgemm(hndl, transA, transB, A.d_row, B.d_column, A.d_column,
        	descrA, A.d_nnz, A.d_csrVal, A.d_csrRowPtr, A.d_csrColInd,
        	descrB, B.d_nnz,B.d_csrVal, B.d_csrRowPtr, B.d_csrColInd,
        	descrC, C->d_csrVal, C->d_csrRowPtr, C->d_csrColInd);

 	CUSPARSE_CHECK(stat);

	return 0;
}




